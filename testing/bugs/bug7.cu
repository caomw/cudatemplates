
#include <hip/hip_runtime.h>
/*
  compiling this program with "nvcc -c bug7.cu" gives the following message:

  ### Assertion failure at line 123 of ../../be/cg/NVISA/expand.cxx:
  ### Compiler Error in file /tmp/tmpxft_00006ce7_00000000-7_bug7.cpp3.i during Code_Expansion phase:
  ### unexpected mtype
  nvopencc INTERNAL ERROR: /usr/open64/lib//be returned non-zero status 1

  Distribution: openSUSE-11.1
  Architecture: x86_64
  gcc-4.3.2
  CUDA toolkit: 2.3beta
*/

__device__ void function()
{
  float3 x0;
  float3 array[8];
  array[0] = x0;
  int i = 0;

  for(;;) {
    if(array[i].x < 0) {
      i++;
      continue;
    }

    array[i] = x0;
  }
}

__global__ void kernel()
{
  function();
}
